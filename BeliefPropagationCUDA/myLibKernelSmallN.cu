#include "hip/hip_runtime.h"
#ifndef MYLIBKERNELSMALLN_CU
#define MYLIBKERNELSMALLN_CU

#include <limits.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "myLibKernelSmallN.h"

#include "myLib.h"

bool debug = false;

void getNumBlocksAndThreadsSmallN(int n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
	Nel primo livello occorre recuperare i dati da g_idata in base agli indici riportati in d_iIndexData
*/
__global__ void
reduce1StepSmallN(double *g_idata, double *g_odata, size_t *d_iIndexData, unsigned int n, unsigned int nArray, bool debug)//, unsigned int fraction)
{	
	// extern serve per rendere l'allocazione della memoria condivisa dinamica 
	// si potrebbe utilizzare quella statica se la dimensione fosse nota a compile time
	// nota: dalla 2.0 in avanti abbiamo 48 KB di shared memory
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
	// legge blocchi di dimensione doppia
	// i scorre tutte le threads di tutti i blocchi pari: blocco0[thread0...threadn] blocco2[thread0...threadn]
//  orig:  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	// in mySum ogni thread dovrebbe avere il valore dell'indice dell'array che deve analizzare 
	// nota: solo metà degli elementi è presa in considerazione, in quanto 0 <= threadIdx.x <= blockDim (non blockDim*2)
    
	//double mySum = (i < n) ? g_idata[d_iIndexData[i]] : 0;
	// ALE
	double mySum;
	if (i < n && d_iIndexData[i] != SIZE_MAX) { //ULONG_MAX) { //SIZE_MAX) {
		mySum = g_idata[d_iIndexData[i]];
	} else {
		mySum = 0;
	}
	//

	//cuPrintf ("CUPRINTF 1- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

	// nella prima passata ogni thread prende come mySum il valore preso all'istruzione precedente 
	// + il valore preso dal blocco successivo (viene così presa in considerazione l'altra metà degli elementi)
    //if (i + (blockDim.x*4) < n)
    //    mySum += g_idata[i+(blockDim.x*4)];
	
	//if (i + blockDim.x < n)
	//	mySum += g_idata[d_iIndexData[i+blockDim.x]];
	// ALE
    if (i + blockDim.x < n && d_iIndexData[i+blockDim.x] != SIZE_MAX) { //ULONG_MAX) { //SIZE_MAX) {
		mySum += g_idata[d_iIndexData[i+blockDim.x]];
	}

    sdata[tid] = mySum;
    __syncthreads();
	//if(tid < 6 || tid > 250) cuPrintf ("CUPRINTF 2- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s >= nArray; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }
	//if((tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- sdata[%d] = %f \n", tid, sdata[tid]);

    // write result for this block to global mem
    //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if(tid < nArray){
		g_odata[blockIdx.x*nArray + tid] = sdata[tid]; // orig g_odata[blockIdx.x] = sdata[tid];
		if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);
		}
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
*/
__global__ void
reduce2StepSmallN(double *g_idata, double *g_odata, unsigned int n, unsigned int nArray, bool debug)//, unsigned int fraction)
{	
	// extern serve per rendere l'allocazione della memoria condivisa dinamica 
	// si potrebbe utilizzare quella statica se la dimensione fosse nota a compile time
	// nota: dalla 2.0 in avanti abbiamo 48 KB di shared memory
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
	// legge blocchi di dimensione doppia
	// i scorre tutte le threads di tutti i blocchi pari: blocco0[thread0...threadn] blocco2[thread0...threadn]
//  orig:  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	// in mySum ogni thread dovrebbe avere il valore dell'indice dell'array che deve analizzare 
	// nota: solo metà degli elementi è presa in considerazione, in quanto 0 <= threadIdx.x <= blockDim (non blockDim*2)
    double mySum = (i < n) ? g_idata[i] : 0;
	//cuPrintf ("CUPRINTF 1- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

	// nella prima passata ogni thread prende come mySum il valore preso all'istruzione precedente 
	// + il valore preso dal blocco successivo (viene così presa in considerazione l'altra metà degli elementi)
    //if (i + (blockDim.x*4) < n)
    //    mySum += g_idata[i+(blockDim.x*4)];
    if (i + blockDim.x < n)
        mySum += g_idata[i+blockDim.x];

    sdata[tid] = mySum;
    __syncthreads();
	//if(tid < 6 || tid > 250) cuPrintf ("CUPRINTF 2- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s >= nArray; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }

        __syncthreads();
    }
	//if((tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- sdata[%d] = %f \n", tid, sdata[tid]);

    // write result for this block to global mem
    //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if(tid < nArray){
		g_odata[blockIdx.x*nArray + tid] = sdata[tid]; // orig g_odata[blockIdx.x] = sdata[tid];
		if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);
		}
}

double *reduceSmallNArray(int  n,
				  	int nArray,
                 	int  numThreads,
                  	int  numBlocks,
                  	int  maxThreads,
                  	int  cpuFinalThreshold,
                  	double *h_odata,
                  	double *d_idata,
                  	double *d_odata,
					size_t *d_iIndexData) //, int nArrayVero)
{
	bool needReadBack = true;
	
    hipDeviceSynchronize();
    	// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
    	dim3 dimBlock(numThreads, 1, 1);
    	// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
	    dim3 dimGrid(numBlocks, 1, 1);
		// when there is only one warp per block, we need to allocate two warps
    	// worth of shared memory so that we don't index shared memory out of bounds
    	int smemSize = (numThreads <= 32) ? 2 * numThreads * sizeof(double) : numThreads * sizeof(double);

		hipError_t error;
		
		unsigned int dimResult = nArray * sizeof(double);
		//unsigned int dimResult = nArrayVero * sizeof(double);
		double *gpu_result = (double *) malloc(dimResult);
        for (int i=0; i < nArray; i++) gpu_result[i]=0;	
		//for (int i=0; i < nArrayVero; i++) gpu_result[i]=0;

			if(debug) { assert (nArray%dimBlock.x == 0); cudaPrintfInit();
					// execute the kernel
					printf("\n1 esecuzione:\n %d dimGrid\n %d dimBlock\n %d smemSize\n",dimGrid.x, dimBlock.x, smemSize ); }
			reduce1StepSmallN<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, debug);//, fraction);
			if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
        
        if (true)
        {
	        int s=numBlocks;

            // sum partial sums from each block on CPU
            // copy result from device to host
              hipMemcpy(h_odata, d_odata, s * sizeof(double) * nArray, hipMemcpyDeviceToHost);
			  error = hipGetLastError();
			  if(error != hipSuccess)
			  {
			    // print the CUDA error message and exit
			    printf("\nCUDA error1: %s\n", hipGetErrorString(error));
			    exit(-1);
			  }	

			double prec;
              for (int i=0; i < nArray; i++) 
			//for (int i=0; i < nArrayVero; i++) 
			{
					for(int j=0; j < s; j++) {
						gpu_result[i] += h_odata[i*s+j];
					}
					prec=gpu_result[0];
					if(debug && (prec != gpu_result[i])) {
						printf("\n CPU ERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); 
						printf("\n CPU          gpu_result[%d]=%f     s = %d, i = %d", i, gpu_result[i], s, i);
					}
					prec=gpu_result[i];
              }
			if(debug) { 	printf("\n CPU gpu_result[0]=%f     s = %d", gpu_result[0], s); }

            needReadBack = false;
        }
        else
        {
            // sum partial block sums on GPU
            int s=numBlocks;

            while (s > cpuFinalThreshold)
            {
                int threads = 0, blocks = 0;
				int nElements = s*nArray;

				// le righe seguenti si potrebbero accorpare in una funzione
                getNumBlocksAndThreadsSmallN(nElements, maxThreads, blocks, threads);	//forse è 2*s //orig:s
				dim3 dimBlock(threads, 1, 1);
				dim3 dimGrid(blocks, 1, 1);
				// when there is only one warp per block, we need to allocate two warps
			    // worth of shared memory so that we don't index shared memory out of bounds
				smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

				// do in input al reduce da rielaborare gli stessi dati
				if(debug) { cudaPrintfInit();
					printf("\n\t iter esecuzione:\n %d dimGrid\n %d dimBlock\n %d smemSize\n %d s\n %d n\n",dimGrid.x, dimBlock.x, smemSize, s, n);
					printf("nElements: %d \n",nElements); }
				reduce2StepSmallN<<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray, debug);	
														// con i numeri piccoli funziona con nElements=n*nArray
				if(debug) { cudaPrintfDisplay (stdout, true);   cudaPrintfEnd (); }
				s = blocks; // orig: s = (s + (threads*2-1)) / (threads*2);
            }
			if(debug) { printf("\n passato il while... s = %d",s); }
            if (s > 1)
            {
                // copy result from device to host
				if(debug) { printf("\n s = %d, nArray = %d",s,nArray);}
                hipMemcpy(h_odata, d_odata, s * sizeof(double) * nArray, hipMemcpyDeviceToHost);
			   error = hipGetLastError();
			  if(error != hipSuccess)
			  {
			    // print the CUDA error message and exit
			    printf("\nCUDA error1: %s\n", hipGetErrorString(error));
			    exit(-1);
			  }	

				double prec;
                for (int i=0; i < nArray; i++) 
				//for (int i=0; i < nArrayVero; i++)
                {
					for(int j=0; j < s; j++) {
						gpu_result[i] += h_odata[i*s+j];
						//printf("\n FINAL h_odata[%d*%d+%d]=%f", i,s,j, h_odata[i*s+j]); 
					}
					prec=gpu_result[0];
					if(debug && (prec != gpu_result[i])) {
						printf("\n GPU ERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); 
						printf("\n GPU          gpu_result[%d]=%f     s = %d, i = %d", i, gpu_result[i], s, i);
					}
					prec=gpu_result[i];
                }
				if(debug) { printf("\n GPU FINAL gpu_result[0]=%f     s = %d", gpu_result[0], s); }
                needReadBack = false;
           }
}
		if (needReadBack)
	    {
	        // copy final sum from device to host
          hipMemcpy(gpu_result, d_odata, sizeof(double)*nArray, hipMemcpyDeviceToHost);
			//hipMemcpy(gpu_result, d_odata, sizeof(double)*nArrayVero, hipMemcpyDeviceToHost);
		  hipError_t error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error2: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
	    }
	return gpu_result;   
}

double * runSmallN(int size, int nArray, double *h_idata, size_t *h_iIndexData) { //, int nArrayVERO){
	//int size = 1<<24;    // number of elements to reduce -> default:  16777216						// ALE
	//int nArray = 1024;	// m = 16384 																// ALE
	//size = 4096;																					// ALE
	//nArray = 1;																						// ALE
    int maxThreads = 256;  // number of threads per block
    int cpuFinalThreshold = 1;
    printf("%d elements\n", size);
    printf("%d nArray\n", nArray);
    printf("%d maxThreads\n", maxThreads);

        // create input
        unsigned int bytesValuesInput = size * sizeof(double);										// ALE
        //double *h_idata = (double *) malloc(bytesValuesInput);										// ALE
		unsigned int bytesIndexInput = size * sizeof(double);										// ALE
		//int *h_iIndexData = (int *) malloc(bytesIndexInput);										// ALE
		
		//for (int i=0; i<size; i++) { h_idata[i] = 1; h_iIndexData[i]=i; }							// ALE
		
        int numBlocks = 0;
        int numThreads = 0;
		getNumBlocksAndThreadsSmallN(size, maxThreads, numBlocks, numThreads);
        
	    if (numBlocks == 1) cpuFinalThreshold = 1;
        
        // allocate mem for the result on host side
		unsigned int bytesOutput = sizeof(double)*nArray*numBlocks;
        double *h_odata = (double *) malloc(bytesOutput);
        printf("%d blocks\n\n", numBlocks);
        
		// allocate device memory and data
        double *d_idata = NULL;
		size_t *d_iIndexData = NULL;
        double *d_odata = NULL;
        hipMalloc((void **) &d_idata, bytesValuesInput);
        hipMalloc((void **) &d_iIndexData, bytesIndexInput);
        hipMalloc((void **) &d_odata, bytesOutput);

        // copy data directly to device memory
        hipMemcpy(d_idata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);
        hipMemcpy(d_iIndexData, h_iIndexData, bytesIndexInput, hipMemcpyHostToDevice);
	    printf("\nbytesOutput=%d\n",bytesOutput);
        hipMemcpy(d_odata, h_idata, bytesOutput, hipMemcpyHostToDevice);

		// per i test si può inserire un warm-up

		hipEvent_t start, stop;
		float time;	// GPU TIME
		hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
		hipEventCreate(&stop);	
		hipEventRecord( start, 0 );
		
        double *gpu_result;																		// ALE è lungo nArray

        gpu_result = reduceSmallNArray(size, nArray, numThreads, numBlocks, maxThreads,
			cpuFinalThreshold, h_odata, d_idata, d_odata, d_iIndexData); //, nArrayVERO);

		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &time, start, stop );
		printf ("\nTime for the kernel: %f ms\n", time);											// ALE 
		double prec=gpu_result[0];
		for(int i=0; i < nArray ; i++)	{
			if(prec != gpu_result[i]) printf("\nERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]);
			prec=gpu_result[i];
		//    printf("\nGPU result = %f\n", gpu_result[i]);
		}
        printf("\nGPU result       \t= \t %f\n", prec);
		assert ((size % nArray) == 0);
		printf("risultato atteso \t= \t %d\n", size / nArray);
	    
        //free(h_idata);
        //free(h_odata);
		hipFree(d_idata);
        hipFree(d_odata);
		hipEventDestroy( start );
		hipEventDestroy( stop );
		
		return gpu_result;
}

void selectGPU()
{
	hipDeviceProp_t deviceProp;
    deviceProp.major = 1;

		// da fare nel caso dei double (perchè necessitano di maggiori risorse), altrimenti sarebbero state settate a 1 e 10
	    deviceProp.minor = 3;
	    int minimumComputeVersion = 13;

	    int dev = findCudaDevice();

	    hipGetDeviceProperties(&deviceProp, dev);

	    if ((deviceProp.major * 10 + deviceProp.minor) >= minimumComputeVersion)
	    {
	        printf("Using Device %d: %s\n\n", dev, deviceProp.name);
	        hipSetDevice(dev);
	    }
	    else
	    {
	        printf("Error: the selected device does not support the minimum compute capability of %d.%d.\n\n",
	               minimumComputeVersion / 10, minimumComputeVersion % 10);

	        hipDeviceReset();
	        exit(EXIT_FAILURE);
	    }
}

void resetGPU()
{
	hipDeviceReset();
}

#endif

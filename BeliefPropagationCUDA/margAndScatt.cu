#include "hip/hip_runtime.h"
#ifndef MARGANDSCATT_CU
#define MARGANDSCATT_CU

#include "../BeliefPropagationCpp/Config.h"

#include <limits.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "margAndScatt.h"

#include "myLib.h"

#if NORMALIZZA_AD_OGNI_PASSO
#include "hipblas.h"
#endif

#ifndef DEBUG_FLAG
//const bool debug = true; //false;
#define DEBUG_FLAG FALSE
#endif

/*
#ifndef SIZE_MAX
#define SIZE_MAX ((size_t)-1)
#endif
*/


//////////////////////////////////////////// RIDUZIONE ==> MARGINALIZATION //////////////////////////////////////////////////////

//////////////////////////////////////////// SMALL //////////////////////////////////////////////////////
void getNumBlocksAndThreadsSmallN(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
	Nel primo livello occorre recuperare i dati da g_idata in base agli indici riportati in d_iIndexData
*/
template <unsigned int blockSize> // forse basta usare blockDim.x
__global__ void
reduce1StepSmallN(double *g_idata, double *g_odata, size_t *d_iIndexData, size_t n, size_t const nArray, size_t dimDataTable)
{	
	// extern serve per rendere l'allocazione della memoria condivisa dinamica 
	// si potrebbe utilizzare quella statica se la dimensione fosse nota a compile time
	// nota: dalla 2.0 in avanti abbiamo 48 KB di shared memory
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    size_t tid = threadIdx.x;
	// legge blocchi di dimensione doppia
	// i scorre tutte le threads di tutti i blocchi pari: blocco0[thread0...threadn] blocco2[thread0...threadn]
//  orig:  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	size_t i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	// in mySum ogni thread dovrebbe avere il valore dell'indice dell'array che deve analizzare 
	// nota: solo metà degli elementi è presa in considerazione, in quanto 0 <= threadIdx.x <= blockDim (non blockDim*2)
	
	size_t index1 = d_iIndexData[i+blockDim.x];				// DA INSERIRE NELLA VERSIONE ROLLED
	size_t index2 = d_iIndexData[i];
	
    //double mySum = (i < n && index2 != SIZE_MAX) ? g_idata[index2] : 0;
	double mySum = (i < n && index2 < dimDataTable) ? g_idata[index2] : 0;
	//cuPrintf ("CUPRINTF 1- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

	// nella prima passata ogni thread prende come mySum il valore preso all'istruzione precedente 
	// + il valore preso dal blocco successivo (viene così presa in considerazione l'altra metà degli elementi)
    //if (i + (blockDim.x*4) < n)
    //    mySum += g_idata[i+(blockDim.x*4)];
    //if (i + blockDim.x < n && index1 != SIZE_MAX)
	if (i + blockDim.x < n && index1 < dimDataTable)
        mySum += g_idata[index1];

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem			32*16=512		(nArray*32)  -> nArray*(1<<2) -> 64   (32/nArray)
	switch(nArray){
	case 1: for (size_t s=blockDim.x/2; s >= nArray*64; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 2: for (size_t s=blockDim.x/2; s >= nArray*32; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 4: for (size_t s=blockDim.x/2; s >= nArray*16; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 8: for (size_t s=blockDim.x/2; s >= nArray*8; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 16: for (size_t s=blockDim.x/2; s >= nArray*4; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 32: for (size_t s=blockDim.x/2; s >= nArray*2; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	default: for (size_t s=blockDim.x/2; s >= nArray; s>>=1){								// ALE LAST CORR
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();

		} break;
		}	
    /*for (unsigned int s=blockDim.x/2; s >= nArray*2; s>>=1) // orig: s >= nArray
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
		if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 2- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);


        __syncthreads();
    }*/
	
	
	

#if DEBUG_FLAG
	if( //debug && 
	   (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

	if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double *smem = sdata;
    	    if (blockSize >=  64 && nArray < 64) smem[tid] = mySum = mySum + smem[tid + 32]; 

			#if DEBUG_FLAG
			if(//debug && 
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 4- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif


			if (blockSize >=  32 && nArray < 32) smem[tid] = mySum = mySum + smem[tid + 16];
			
#if DEBUG_FLAG
			if(//debug && 
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 5- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

			
			if (blockSize >=  16 && nArray < 16) smem[tid] = mySum = mySum + smem[tid + 8];

			#if DEBUG_FLAG
			if(//debug && 
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 6- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

			
	        if (blockSize >=  8 && nArray < 8) smem[tid] = mySum = mySum + smem[tid + 4];

			#if DEBUG_FLAG
			if(//debug && 
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 7- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

	
	        if (blockSize >=  4 && nArray < 4) smem[tid] = mySum = mySum + smem[tid + 2];

			#if DEBUG_FLAG
			if(//debug &&
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 8- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

	
	        if (blockSize >=  2 && nArray < 2) smem[tid] = mySum = mySum + smem[tid + 1];

			#if DEBUG_FLAG
			if(//debug && 
				(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 9- sdata[%d] = %f mySum = %f \n", tid, sdata[tid], mySum);
#endif

	
	    }



    // write result for this block to global mem
    //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	if(tid < nArray){
		g_odata[blockIdx.x*nArray + tid] = sdata[tid]; // orig g_odata[blockIdx.x] = sdata[tid];
		//if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);	
		cuPrintf ("CUPRINTF 10- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);
		}
}

/*
    This version uses n/2 threads --
    - performs the first level of reduction when reading from global memory.
    - unrolls the last warp to avoid synchronization where it
    isn't needed.

Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
If blockSize > 32, allocate blockSize*sizeof(T) bytes.

*/
template <unsigned int blockSize> // forse basta usare blockDim.x
__global__ void
reduce2StepSmallN(double *g_idata, double *g_odata, size_t n, size_t const nArray)
{	
	// extern serve per rendere l'allocazione della memoria condivisa dinamica 
	// si potrebbe utilizzare quella statica se la dimensione fosse nota a compile time
	// nota: dalla 2.0 in avanti abbiamo 48 KB di shared memory
    extern __shared__ double sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    size_t tid = threadIdx.x;
	// legge blocchi di dimensione doppia
	// i scorre tutte le threads di tutti i blocchi pari: blocco0[thread0...threadn] blocco2[thread0...threadn]
//  orig:  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	size_t i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	// in mySum ogni thread dovrebbe avere il valore dell'indice dell'array che deve analizzare 
	// nota: solo metà degli elementi è presa in considerazione, in quanto 0 <= threadIdx.x <= blockDim (non blockDim*2)
    double mySum = (i < n) ? g_idata[i] : 0;
	//cuPrintf ("CUPRINTF 1- blockIdx.x = %d mySum = %f \n",blockIdx.x, mySum);

	// nella prima passata ogni thread prende come mySum il valore preso all'istruzione precedente 
	// + il valore preso dal blocco successivo (viene così presa in considerazione l'altra metà degli elementi)
    if (i + blockDim.x < n)
        mySum += g_idata[i+blockDim.x];

    sdata[tid] = mySum;
    __syncthreads();

	switch(nArray){
	case 1: for (size_t s=blockDim.x/2; s >= nArray*64; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 2: for (size_t s=blockDim.x/2; s >= nArray*32; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 4: for (size_t s=blockDim.x/2; s >= nArray*16; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 8: for (size_t s=blockDim.x/2; s >= nArray*8; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 16: for (size_t s=blockDim.x/2; s >= nArray*4; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	case 32: for (size_t s=blockDim.x/2; s >= nArray*2; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();
	    
		} break;
	default: for (size_t s=blockDim.x/2; s >= nArray; s>>=1){
		if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }    __syncthreads();

		} break;
		}

    // do reduction in shared mem
    /*for (unsigned int s=blockDim.x/2; s >= (1<<(32/nArray)); s>>=1) // orig: s >= nArray
    {
        if (tid < s)
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
		if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("\nCUPRINTF 1 - ciclo\n");
		
		
        __syncthreads();
    }*/
	//if((tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- sdata[%d] = %f \n", tid, sdata[tid]);

#if DEBUG_FLAG
    if(//debug && 
		(tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 12 - mySum = %f\n", mySum);
#endif

	if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile double *smem = sdata;
    	    if (blockSize >=  64 && nArray < 64) smem[tid] = mySum = mySum + smem[tid + 32]; 
			if (blockSize >=  32 && nArray < 32) smem[tid] = mySum = mySum + smem[tid +  16];
			if (blockSize >=  16 && nArray < 16) smem[tid] = mySum = mySum + smem[tid + 8];
	        if (blockSize >=  8 && nArray < 8) smem[tid] = mySum = mySum + smem[tid + 4];
	        if (blockSize >=  4 && nArray < 4) smem[tid] = mySum = mySum + smem[tid + 2];
	        if (blockSize >=  2 && nArray < 2) smem[tid] = mySum = mySum + smem[tid + 1];
	
	
	    }



	if(tid < nArray){
		g_odata[blockIdx.x*nArray + tid] = sdata[tid]; // orig g_odata[blockIdx.x] = sdata[tid];
		//if(debug && (tid == 0 || tid == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);	
		cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, tid, g_odata[tid]);
		}
}

void reduceSmallNArray(size_t  n,
				  	size_t nArray,
                 	int  numThreads,
                  	int  numBlocks,
                  	int  maxThreads,
                  	int  cpuFinalThreshold,
                  	//double *h_odata,
                  	double *d_idata,
                  	double *d_odata,
					size_t *d_iIndexData,
					size_t dimRisultato,
					size_t sizeDataTable
					)
{
	//bool needReadBack = true;
	
    hipDeviceSynchronize();
    	// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
    	dim3 dimBlock(numThreads, 1, 1);
    	// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
	    dim3 dimGrid(numBlocks, 1, 1);
		// when there is only one warp per block, we need to allocate two warps
    	// worth of shared memory so that we don't index shared memory out of bounds
    	int smemSize = (numThreads <= 32) ? 2 * numThreads * sizeof(double) : numThreads * sizeof(double);

		hipError_t error;
		
		//size_t dimResult = dimRisultato * sizeof(double);
		/* 
		double *gpu_result = (double *) malloc(dimResult);
		for (size_t i=0; i < nArray; i++) gpu_result[i]=0;
		*/

		#if DEBUG_FLAG
			//if(debug) { 
				cudaPrintfInit();
					// execute the kernel
					printf("\n1 esecuzione:\n %d dimGrid\n %d dimBlock\n %d smemSize\n",dimGrid.x, dimBlock.x, smemSize ); 
			//}
#endif

			//reduce1StepSmallN<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, debug);

			//printf("numThreads: %d", numThreads);
					switch (numThreads){
						case 512:
						reduce1StepSmallN<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 256:
						reduce1StepSmallN<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 128:
						reduce1StepSmallN<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 64:
						reduce1StepSmallN<64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 32:
						reduce1StepSmallN<32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 16:
						reduce1StepSmallN<16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 8:
						reduce1StepSmallN<8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 4:
						reduce1StepSmallN<4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 2:
						reduce1StepSmallN<2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
						case 1:
						reduce1StepSmallN<1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, d_iIndexData, n, nArray, sizeDataTable);	break;
					}
			

			#if DEBUG_FLAG
			//if(debug) { 
				cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); 
			//}
			#endif

        
            // sum partial block sums on GPU
            int s=numBlocks;
			int iterDebug = 0;
            
			while (s > cpuFinalThreshold) // abbiamo visto che cpuFinalThreshold è 1
            {
                int threads = 0, blocks = 0;
				size_t nElements = s*nArray;

                getNumBlocksAndThreadsSmallN(nElements, maxThreads, blocks, threads);
				dim3 dimBlock(threads, 1, 1);
				dim3 dimGrid(blocks, 1, 1);
				// when there is only one warp per block, we need to allocate two warps
			    // worth of shared memory so that we don't index shared memory out of bounds
				smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

				// do in input al reduce da rielaborare gli stessi dati

				#if DEBUG_FLAG
				//if(debug) { 
					cudaPrintfInit();
					printf("\n\t iter esecuzione:\n %d dimGrid\n %d dimBlock\n %d smemSize\n %d s\n %d n\n",dimGrid.x, dimBlock.x, smemSize, s, n);
					printf("nElements: %d \n",nElements); 
					//}
				#endif


				switch (threads){
					case 512:
					reduce2StepSmallN<512><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 256:
					reduce2StepSmallN<256><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 128:
					reduce2StepSmallN<128><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 64:
					reduce2StepSmallN<64><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 32:
					reduce2StepSmallN<32><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 16:
					reduce2StepSmallN<16><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 8:
					reduce2StepSmallN<8><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 4:
					reduce2StepSmallN<4><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 2:
					reduce2StepSmallN<2><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
					case 1:
					reduce2StepSmallN<1><<< dimGrid, dimBlock, smemSize >>>(d_odata, d_odata, nElements, nArray);	break;
				}
					
/*
				#if DEBUG_FLAG
				//if(debug) { 
					cudaPrintfDisplay (stdout, true);   cudaPrintfEnd (); 
				//}
				#endif
*/
				
				error = hipGetLastError();
			  	if(error != hipSuccess)
			  	{
			    // print the CUDA error message and exit
			    printf("\nCUDA kernel error: %s\n", hipGetErrorString(error));
			    exit(-1);
			  }
				s = blocks;
				iterDebug++;
            }
/*
			#if DEBUG_FLAG
			//if(debug) { 
				printf("\n passato il while... s = %d",s); printf("\niterazioni: %d",iterDebug);
			//}
			#endif
*/
			//if(iterDebug!=0) { printf("\n ci sono iterazioni");exit(-1);}
            /*
			if (s > 1)
            {
                // copy result from device to host
				if(debug) { printf("\n s = %d, nArray = %d",s,nArray);}
                //hipMemcpy(h_odata, d_odata, s * sizeof(double) * nArray, hipMemcpyDeviceToHost);
				 hipMemcpy(h_odata, d_odata, s * sizeof(double) * dimRisultato, hipMemcpyDeviceToHost);
			   error = hipGetLastError();
			  if(error != hipSuccess)
			  {
			    // print the CUDA error message and exit
			    printf("\nCUDA error1: %s\n", hipGetErrorString(error));
			    exit(-1);
			  }	

				double prec;
                //for (int i=0; i < nArray; i++) 
				for (size_t i=0; i < dimRisultato; i++) 
                {
					for(int j=0; j < s; j++) {
						gpu_result[i] += h_odata[i*s+j];
					}
					prec=gpu_result[0];
					if(debug && (prec != gpu_result[i])) {
						printf("\n GPU ERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); 
						printf("\n GPU          gpu_result[%d]=%f     s = %d, i = %d", i, gpu_result[i], s, i);
					}
					prec=gpu_result[i];
                }
				if(debug) { printf("\n GPU FINAL gpu_result[0]=%f     s = %d", gpu_result[0], s); }
                needReadBack = false;
           }
		   */
		/*
		if (needReadBack)
	    {
	        // copy final sum from device to host
          // non devo copiarla e restituisco direttamente il puntatore in GPU
		 
		 hipMemcpy(gpu_result, d_odata, sizeof(double)*nArray, hipMemcpyDeviceToHost);
			// in teoria hipError_t si può togliere perchè è dichiarato su
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error2: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
		  
	    }
	return gpu_result;  
	*/
}

double* marginalizationSmallN(size_t size, size_t nArray, double *h_idata, size_t *h_iIndexData, size_t dimInput, size_t dimRisultato
							  , long long* elapsedSum, long long* elapsedDivMul
							  ){
	// TESTA CON NVPROF
	//int size = 1<<24; // 1<<24;    // number of elements to reduce -> default:  16777216				// ALE
	//int nArray = 1<<5;	// m = 16384 
	//int m = 1<<6; // 4 7 10 14 17 20 
	//da testare con int nArray = 1<<4;1<<9;
	//int m = 1<<19; 1<<15;
	//size = nArray*m;	
	size_t m = size / nArray;
	//size = 1<<16;																						// ALE
	//nArray = 1<<20;																					// ALE
    //int maxThreads = 512;				// number of threads per block
    int cpuFinalThreshold = 1;				// da testare
    /* STAMPE */
/*
#if DEBUG_FLAG
	printf("%d elements\n", size);
    printf("%d nArray\n", nArray);
    printf("%d m\n", m);
	printf("%d dimCriccaDaSommare\n", dimInput);
	printf("%d dimSeparatore\n", dimRisultato);
#endif
*/
    //printf("%d maxThreads\n", maxThreads);
	/* */

	if (m == 1) {
		// ALE 2014-02-23
		/*
		gpu_result = (double *) malloc(dimRisultato * sizeof(double));
		for (size_t i = 0; i < dimInput; i++)
			gpu_result[i] = h_idata[i];
		return gpu_result;
		*/
		double *d_odata = NULL;
		size_t bytesValuesInput = sizeof(double)*dimInput;
		hipMalloc((void **) &d_odata, bytesValuesInput);
		// copy data directly to device memory
        hipMemcpy(d_odata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);
		return d_odata;
	}


	//for(int maxThreads = 128; maxThreads < 1024; maxThreads++){
	/*	float bestAverageTime = 100.0;
		int bestcpuFinalThreshold=1;
		int bestMaxThreads=32; */
		
	//for(int cpuFinalThreshold = 1; cpuFinalThreshold <200; cpuFinalThreshold=cpuFinalThreshold+10){
		//for(int maxThreads = nArray; maxThreads <1024; maxThreads*=2){
		int maxThreads = 512;

			//float total = 0.0;
			//for(int i =0; i<100; i++){
					//float time=0;	// GPU TIME
			        // create input
			        size_t bytesValuesInput = dimInput * sizeof(double);										// ALE
			        //double *h_idata = (double *) malloc(bytesValuesInput);										// ALE
					size_t bytesIndexInput = size * sizeof(size_t);											// ALE
					//int *h_iIndexData = (int *) malloc(bytesIndexInput);										// ALE
		
					//for (int i=0; i<size; i++) { h_idata[i] = 1; h_iIndexData[i]=i; }							// ALE
		
		
	        		int numBlocks = 0;
			        int numThreads = 0;
					getNumBlocksAndThreadsSmallN(size, maxThreads, numBlocks, numThreads);

					if (numBlocks == 1) { cpuFinalThreshold = 1; 
/*
						#if DEBUG_FLAG
						//if(debug) 
							printf("cpuFinalThreshold = 1\n");
						#endif
*/
					}
        
			        // allocate mem for the result on host side
					size_t bytesOutput = sizeof(double)*nArray*numBlocks;
			        //double *h_odata = (double *) malloc(bytesOutput);
/*
					#if DEBUG_FLAG
					//if(debug) 
						printf("%d blocks\n", numBlocks);
					#endif
*/
        
					// allocate device memory and data
			        double *d_idata = NULL;
					size_t *d_iIndexData = NULL;
			        double *d_odata = NULL;
			        hipMalloc((void **) &d_idata, bytesValuesInput);
			        hipMalloc((void **) &d_iIndexData, bytesIndexInput);
			        hipMalloc((void **) &d_odata, bytesOutput);

			        // copy data directly to device memory
			        hipMemcpy(d_idata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);
			        hipMemcpy(d_iIndexData, h_iIndexData, bytesIndexInput, hipMemcpyHostToDevice);
				    //printf("\nbytesOutput=%d\n",bytesOutput);
			        //hipMemcpy(d_odata, h_idata, bytesOutput, hipMemcpyHostToDevice);			//NON SERVE!!!

#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
					// per i test si può inserire un warm-up
					hipEvent_t start, stop;
					hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
					hipEventCreate(&stop);	
					hipEventRecord( start, 0 );
#endif
			        //double *gpu_result;																		// ALE è lungo nArray

/*
#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
	std::chrono::system_clock::time_point begin = std::chrono::high_resolution_clock::now();
#endif
*/
			        //gpu_result = 
					reduceSmallNArray(size, nArray, numThreads, numBlocks, maxThreads,
			                                        cpuFinalThreshold, d_idata, d_odata, d_iIndexData, dimRisultato, dimInput);
/*
#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
	std::chrono::system_clock::time_point end = std::chrono::high_resolution_clock::now();
	*elapsedSum += std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
#endif
*/
#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
					hipEventRecord( stop, 0 );
					hipEventSynchronize( stop );
					hipEventElapsedTime( &time, start, stop );
					//total+=time;										// ALE 
					*elapsedSum += time * 1000000;
#endif
					//double prec=gpu_result[0];
					/*for(int i=0; i < nArray ; i++)	{
						if(prec != gpu_result[i]) printf("\nERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f\n",i,prec,i, gpu_result[i]);
						prec=gpu_result[i];
					//    printf("\nGPU result = %f\n", gpu_result[i]);
					}*/
					//assert (m==prec);
					//if(prec != m) printf("\nERRORE:  m=%d != prec=%f\n",m,prec);
#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
					hipEventDestroy( start );													//ALE
					hipEventDestroy( stop );
#endif													//ALE
		
		
			        //printf("\nGPU result       \t= \t %f\n", prec);
					assert ((size % nArray) == 0);
					//printf("risultato atteso \t= \t %d\n", size / nArray);

					//free(gpu_result);	    
			        //free(h_idata);													//ALE
					//free(h_iIndexData);
			        //free(h_odata);													//ALE
					hipFree(d_idata);													//ALE
			        //hipFree(d_odata);													//ALE
					hipFree(d_iIndexData);
			//}
			/*
			if(bestAverageTime > (total/100) && (!debug)) { bestAverageTime = (total/100); bestMaxThreads = maxThreads;
			// printf("partialBestAverageTime: %f\n bestMaxThreads = %d\n", bestAverageTime, bestMaxThreads); 
			}
			*/
		//}
		//if(!debug)printf("bestAverageTime: %f\n bestMaxThreads = %d\n", bestAverageTime, bestMaxThreads);
			return d_odata;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////// BIG //////////////////////////////////////////////////////

void getNumBlocksAndThreadsBigN(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}

__global__ void
reduce1StepBigN(double *g_idata, double *g_odata, size_t *d_iIndexData, size_t n, size_t halfN, size_t nArray, size_t dimDataTable)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;

	// somma elementi della prima metà con elementi della seconda metà
	int index1 = d_iIndexData[i+halfN];
	int index2 = d_iIndexData[i];
		
    if (i < halfN){ // sta sempre dentro all'if
		//g_odata[i] = g_idata[d_iIndexData[i]] + g_idata[d_iIndexData[i+halfN]];
		//double mySum = (index1 != SIZE_MAX) ? g_idata[index1] : 0;
		double mySum = (index1 < dimDataTable) ? g_idata[index1] : 0;
		//g_odata[i] = (index2 != SIZE_MAX) ? g_idata[index2] + mySum : mySum;
		g_odata[i] = (index2 < dimDataTable) ? g_idata[index2] + mySum : mySum;
		#if DEBUG_FLAG
			if(//debug && 
				(threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
		#endif
	}
}

__global__ void														// non so se passarlo come parametro o farlo calcolare alle theads... TESTARE
reduce2StepBigN(double *g_idata, double *g_odata, size_t n, size_t halfN, size_t nArray)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	unsigned const int i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;
    //double mySum = (i < half) ? g_idata[i] : 0;
	// somma elementi della prima metà con elementi della seconda metà
    if (i < halfN){ // sta sempre dentro all'if
        //mySum += g_idata[i+blockDim.x];
		//double mySum = g_idata[i] + g_idata[i+blockDim.x];
		g_odata[i] = g_idata[i] + g_idata[i+halfN];
/*
		#if DEBUG_FLAG
			if(//debug && 
				(threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
		#endif
*/
	}
}



void reduceBigNArray(size_t  n,
				  	size_t nArray,
                 	int  numThreads,
                  	int  numBlocks,
                  	int  maxThreads,
                  	//int  cpuFinalThreshold,
                  	double *h_odata, // si può togliere ??
                  	double *d_idata,
                  	double *d_odata,
					size_t *d_iIndexData,
					size_t dimRisultato,
					size_t sizeDataTable
					)
{
	//bool needReadBack = true;
	
    hipDeviceSynchronize();
    	// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
    	dim3 dimBlock(numThreads, 1, 1);
    	// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
	    dim3 dimGrid(numBlocks, 1, 1);

		//hipError_t error;
		
		//size_t dimResult = nArray * sizeof(double);
		//size_t dimResult = dimRisultato * sizeof(double);

		// ALE 2014-02-23
		/*
		double *gpu_result = (double *) malloc(dimResult);
		// ALE_DOMANDA
		// SERVE??? non viene sovrascritto poi dalla cudaMemCpy???
		for (int i=0; i < dimRisultato; i++) gpu_result[i]=0;		
		//
		*/
/*
		#if DEBUG_FLAG
		//if(debug) { 
			cudaPrintfInit();
				// execute the kernel
				printf("\nSTART esecuzione:\n %d dimGrid\n %d dimBlock\n",dimGrid.x, dimBlock.x ); 
		//}
		#endif
*/

		reduce1StepBigN<<< dimGrid, dimBlock>>>(d_idata, d_odata, d_iIndexData, n, n>>1, nArray, sizeDataTable);
		
/*
		#if DEBUG_FLAG
		//if(debug) { 
			cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); 
		//}
		#endif
*/
		/*
		unsigned int dimResultCHECK = (n>>1) * sizeof(double);
		double *gpu_resultCHECK = (double *) malloc(dimResultCHECK);
		hipMemcpy(gpu_resultCHECK, d_odata, sizeof(double)*(n>>1), hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error0: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
		double prec=gpu_resultCHECK[0];
		for(int i=0; i < nArray ; i++)	{
			if(prec != gpu_resultCHECK[i]) printf("\n---0 ERRORE:  gpu_resultCHECK[%d-1]=%f != gpu_resultCHECK[%d]=%f",i,prec,i, gpu_resultCHECK[i]);
			prec=gpu_resultCHECK[i];
		//    printf("\nGPU result = %f\n", gpu_result[i]);
		}
		int j=0;
		
        printf("\nSTART %d GPU result       \t= \t %f\n", j, prec);
        */


		for(size_t nElements = n>>1; nElements > nArray; nElements>>=1){
			//sleep(1);
			
			int threads = 0, blocks = 0;
            

			getNumBlocksAndThreadsBigN(nElements, maxThreads, blocks, threads);
			
			dim3 dimBlock(threads, 1, 1);
			dim3 dimGrid(blocks, 1, 1);
			
/*
			#if DEBUG_FLAG
			//if(debug) 
				printf("\n%d elementi:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); 
	
			//if(debug) { 
				cudaPrintfInit();
			
				// execute the kernel
				printf("\n%d esecuzione:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); 
			//}
			#endif
*/

			reduce2StepBigN<<< dimGrid, dimBlock>>>(d_odata, d_odata, nElements, nElements>>1, nArray);
			
/*
			#if DEBUG_FLAG
			//if(debug) { 
				cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); 
			//}
			#endif
*/
			
        }
        /*
		if (needReadBack)
	    {
		*/
	        // copy final sum from device to host
          //hipMemcpy(gpu_result, d_odata, sizeof(double)*nArray, hipMemcpyDeviceToHost);
			
			// ALE CUDA MEM CPY lo faccio dopo visto che lo unisco allo scattering
			/*hipMemcpy(gpu_result, d_odata, dimResult, hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error2: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	*/
	   /*
		}
		*/
		// ALE 2014-02-23
	/*
	return gpu_result;  
	*/

}

//                   dimTabCricca POW2, dimTabSep POW2, tabCricca,     tabIndiciCricca,   dimensione vera tabCricca, dim vera tabSep
double* marginalizationBigN(size_t size, size_t nArray, double *h_idata, size_t *h_iIndexData, size_t dimInput, size_t dimRisultato
							, long long* elapsedSum, long long* elapsedDivMul
							){
	// ALE_GOT
	//size = 1048576;
	//nArray = 16384;
	//

	//int size = 1<<24; // 1<<24;    // number of elements to reduce -> default:  16777216				// ALE
	//int nArray = 1<<4;	// m = 16384 
	//int m = 1<<20; // 4 7 10 14 17 20 
	size_t m = size / nArray;																					// ALE
	//size = 1<<16;																						// ALE
	//nArray = 1<<20;																					// ALE
    int maxThreads = 256;				// number of threads per block
    //int cpuFinalThreshold = 1;
    /*  STAMPE * /
	printf("%d elements\n", size);
    printf("%d nArray\n", nArray);
	printf("%d nArrayInput\n", dimInput);
	printf("%d nArrayRisultato\n", dimRisultato);
    printf("%d m\n", m);
    printf("%d maxThreads\n", maxThreads);
	/ * */

	//double *gpu_result;	

	
	if (m == 1) {
		// ALE 2014-02-23
		/*
		gpu_result = (double *) malloc(dimRisultato * sizeof(double));
		for (size_t i = 0; i < dimInput; i++)
			gpu_result[i] = h_idata[i];
		return gpu_result;
		*/
		double *d_odata = NULL;
		size_t bytesValuesInput = sizeof(double)*dimInput;
		hipMalloc((void **) &d_odata, bytesValuesInput);
		// copy data directly to device memory
        hipMemcpy(d_odata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);
		return d_odata;
	}

	
	//float total = 0.0;

	//for(int i =0; i<100; i++){

        // create input
	size_t bytesValuesInput = dimInput * sizeof(double);										// ALE
        // ALE_GOT
		//double *
		//	h_idata = (double *) malloc(bytesValuesInput);										// ALE
		//
		size_t bytesIndexInput = size * sizeof(size_t);										// ALE
		// ALE_GOT
		//size_t *
		//	h_iIndexData = (size_t *) malloc(bytesIndexInput);										// ALE
		//
		// ALE_GOT
		//for (size_t i=0; i<size; i++) { h_idata[i] = 1; h_iIndexData[i]=i; }							// ALE
		//

       	int numBlocks = 0;
        int numThreads = 0;
		getNumBlocksAndThreadsBigN(size, maxThreads, numBlocks, numThreads);
       
	    //if (numBlocks == 1) cpuFinalThreshold = 1;
       
        // allocate mem for the result on host side
		size_t bytesOutput = sizeof(double)*(size/2);											// ALE
        double *h_odata = (double *) malloc(bytesOutput);
/*
#if DEBUG_FLAG
        if(debug) printf("%d blocks\n", numBlocks);
#endif
*/   
		// allocate device memory and data
        double *d_idata = NULL;
		size_t *d_iIndexData = NULL;
        double *d_odata = NULL;
	
	

        hipMalloc((void **) &d_idata, bytesValuesInput);
        hipMalloc((void **) &d_iIndexData, bytesIndexInput);
        hipMalloc((void **) &d_odata, bytesOutput);

        // copy data directly to device memory
        hipMemcpy(d_idata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);

		// ALE_GOT
		/*
		for (size_t l = 0; l < dimInput; l++)
				printf("%f ", h_idata[l]);
			printf("\n");
			for (size_t l = 0; l < size; l++)
				printf("%d ", h_iIndexData[l]);
		*/
		//	getchar();
		
		//


        hipMemcpy(d_iIndexData, h_iIndexData, bytesIndexInput, hipMemcpyHostToDevice);
	    //printf("\nbytesOutput=%d\n",bytesOutput);
        //hipMemcpy(d_odata, h_idata, bytesOutput, hipMemcpyHostToDevice);			//NON SERVE!!!

		#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
		// per i test si può inserire un warm-up
			hipEvent_t start, stop;
			float time;	// GPU TIME
			hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
			hipEventCreate(&stop);	
		
		
			hipEventRecord( start, 0 );
		#endif

	        //double *gpu_result;																		// ALE è lungo nArray
			
			//ALE 2014-02-23
			/*
	        gpu_result = 
			*/
/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	std::chrono::system_clock::time_point begin = std::chrono::high_resolution_clock::now();
#endif
*/
			reduceBigNArray(size, nArray, numThreads, numBlocks, maxThreads,
							h_odata, d_idata, d_odata, d_iIndexData, dimRisultato, dimInput);
/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	std::chrono::system_clock::time_point end = std::chrono::high_resolution_clock::now();
	*elapsedSum += std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
#endif
*/

	#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &time, start, stop );
			//total+=time; // ALE
			*elapsedSum += time * 1000000;
	#endif

			//printf ("\nTime for the kernel: %f ms\n", time);											// ALE 
			//double prec=gpu_result[0];
			/*for(int i=0; i < nArray ; i++)	{
				if(prec != gpu_result[i]) {printf("\nERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); exit(-1);}
				prec=gpu_result[i];
			//    printf("\nGPU result = %f\n", gpu_result[i]);
			}*/
//			assert (m==prec);

			#if (!TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA)
			hipEventDestroy( start );													// ALE
			hipEventDestroy( stop );													//ALE
			#endif

			
			
	        //printf("\nGPU result       \t= \t %f\n", prec);
			//assert ((size % nArray) == 0);
			//printf("risultato atteso \t= \t %d\n", size / nArray);


			//free(gpu_result);	    
			//free(h_iIndexData);    
	        //free(h_idata);													//ALE
	        //free(h_odata);													//ALE
			hipFree(d_idata);													//ALE
	        //hipFree(d_odata);													//ALE
			hipFree(d_iIndexData);
		//} 
		/*
		printf("average time: %f\n", (total/100));
		*/
		// ALE_GOT
		//return gpu_result;
			return d_odata;
		//


		
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////// MOLTIPLICAZIONI ==> SCATTERING /////////////////////////////////////////////////////

void getNumBlocksAndThreadsDivMult(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = n / threads; //(n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}



// (possibile ottimizzazione: far si che ogni blocco faccia le operazioni sul doppio dei dati
// per avere la coalescenza andrebbe fatto non a livello di thread (th0 legge dato0 e dato1)
// ma th0 legge dato 0 e dato 0+dimBlocco
__global__ void
kernelDivVector(double *g_iVector1Data, double *g_iVector2Data, size_t n)
{	
//	if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 1- \n");
	size_t i = blockIdx.x*(blockDim.x) + threadIdx.x;
	if (i < n)
		// ALE
			g_iVector1Data[i] = (g_iVector2Data[i] > ZERO_DIVISIONE) ? g_iVector1Data[i] / g_iVector2Data[i] : 0; // ZERO_DIVISIONE = 0.0
}

/*void getNumBlocksAndThreadsMultMatrixVector(int n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device; 
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = n/threads; //(n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}*/

	__global__ void
kernelMultMatrixVector(double *d_MatrixData, size_t *d_MatrixIndex, double * g_iVector1Data, size_t dimSeparatorePow2, size_t size, size_t sizeDataTable){ // dimSep, size, dimCricca
	size_t i = blockIdx.x*(blockDim.x) + threadIdx.x;
	size_t index = d_MatrixIndex[i];
	//if (index != SIZE_MAX)
	if (index < sizeDataTable)
		d_MatrixData[index] *= g_iVector1Data[i % dimSeparatorePow2]; // dimSepPow2 vs dimSep ???
	
	//if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- d_MatrixData[%d] = %f \n", index, d_MatrixData[index]);
}

//           dimTabCricca POW2, dimTabSep POW2, tabCricca, tabIndiciCricca,        dimensione vera tabCricca, dim vera tabSep
void scattering(size_t size,  // dimTabCricca POW2
			size_t n, // dimTabSep POW2
			double *iVector1Data, // double *h_iVector2Data qui ci va il device_v1data
			double *h_iVector2Data, // FI
			double *h_MatrixData, // PSI da aggiornare
			size_t *h_MatrixIndex, // Indexig of PSI da aggiornare

			size_t dimCricca, // dimensione vera tabCricca
			size_t dimSeparatore // dim vera tabSep
			, long long* elapsedSum, long long* elapsedDivMul
			, bool isOnHost
			) {
	// TESTA CON NVPROF
	//unsigned int n = 1<<12;//19;	
	//unsigned int m = 1<<12;//5;
	//unsigned int size = m*n;

	

	//int maxThreadsDiv = 512;				/// ATTENZIONE, QUANDO SI FANNO I TEST BISOGNA VARIARE ENTRAMBI I VALORI!!!
	/*
	size_t m = size / n;
	printf("%d n\n", n);
	printf("%d m\n", m);
	printf("%d size\n", size);
	printf("%d dimSep\n", dimSeparatore);
	printf("%d dimCricca\n", dimCricca);
	*/
	hipError_t error;

	//float bestAverageTimeMult = 100.0;
	//float bestAverageTimeDiv = 100.0;
	//unsigned int bestMaxThreadsDiv = 0;
	//unsigned int bestMaxThreadsMult = 0;

	
	//for (int maxThreads = 32; maxThreads < 1024; maxThreads*=2) {
	// ALE
	int maxThreads = 256;

	unsigned int maxThreadsDiv = maxThreads;
	unsigned int maxThreadsMult = maxThreads;

	//float totalDiv = 0.0;
	//float totalMult = 0.0;
		
	//for(int i =0; i<100; i++){
	
		//float timeDiv=0;																						// GPU TIME
		// create input
		// BISOGNA TOGLIERE GLI INDICI DALL'ARRAY, (METTERE L'ARRAY NELLA CONSTANT MEMORY)...
		//size_t bytesVector = n * sizeof(double); // ALE

		// ALE MOD
		size_t bytesVector = dimSeparatore * sizeof(double);

		// FI* (il risultato della divisione lo salva nel g_ di questo)
		// passo direttamente il g_
		//double *h_iVector1Data = (double *) malloc(bytesVector);									// ALE
		// FI
		//double *h_iVector2Data = (double *) malloc(bytesVector);									// ALE

		size_t bytesDataMatrix = dimCricca * sizeof(double);										// ALE
		//double *h_MatrixData = (double *) malloc(bytesDataMatrix);									// ALE
		size_t bytesIndexMatrix = size * sizeof(size_t);									// ALE
		//unsigned int *h_MatrixIndex = (unsigned int *) malloc(bytesIndexMatrix);					// ALE


		/*
		for (unsigned int i=0; i<n; i++) { h_iVector1Data[i] = 10; h_iVector2Data[i] = 5; } 	// risultato atteso divisione: 2	// ALE
		for (unsigned int i=0; i<size; i++) { 
			h_MatrixIndex[i] = i; 
			h_MatrixData[i] = 2; 
		} // risultato atteso moltiplicazione: 4			// ALE
		*/

		int numBlocks = 0;
		int numThreads = 0;
		getNumBlocksAndThreadsDivMult(n, maxThreadsDiv, numBlocks, numThreads);

		//assert(n%numThreads==0);

		// allocate device memory and data
		//double *d_iVector1DataP = NULL;
		double *d_iVector2Data = NULL;
		//hipMalloc((void **) &d_iVector1DataP, bytesVector);
		hipMalloc((void **) &d_iVector2Data, bytesVector);

		// copy data directly to device memory
		//hipMemcpy(d_iVector1Data, h_iVector1Data, bytesVector, hipMemcpyHostToDevice);
		hipMemcpy(d_iVector2Data, h_iVector2Data, bytesVector, hipMemcpyHostToDevice);

		double* d_iVector1Data = NULL;
		if (isOnHost) {
			// devo tirarmelo su dall'host
			hipMalloc((void**) &d_iVector1Data, bytesVector);
			hipMemcpy(d_iVector1Data, iVector1Data, bytesVector, hipMemcpyHostToDevice);
		} else {
			// devo salvarmelo sull'host
			d_iVector1Data = iVector1Data;
			// ALE adesso il valore di h_iVector2Data è sul device e nell'host non mi serve più quindi lo rimpiazzo con d_iVector1Data
			// copy vector from device to host to check errors
			hipMemcpy(h_iVector2Data, d_iVector1Data, bytesVector, hipMemcpyDeviceToHost);
			error = hipGetLastError();
			if(error != hipSuccess)
			{
				// print the CUDA error message and exit
				printf("\nCUDA error12: %s\n", hipGetErrorString(error));
				exit(-1);
			}
			//
		}

		

		hipDeviceSynchronize();
		// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
		dim3 dimBlockDiv(numThreads, 1, 1);
		// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
		dim3 dimGridDiv(numBlocks, 1, 1);

/*
		#if DEBUG_FLAG
		//if(debug) { 
			cudaPrintfInit(); 
			printf("\n DIV:\n %d dimGrid\n %d dimBlock\n",dimGridDiv.x, dimBlockDiv.x ); 
		//}
		#endif
*/


		#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
		hipEvent_t start, stop;
		hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
		hipEventCreate(&stop);	
		hipEventRecord( start, 0 );
		#endif

/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	std::chrono::system_clock::time_point begin = std::chrono::high_resolution_clock::now();
#endif
*/
		kernelDivVector<<< dimGridDiv, dimBlockDiv >>>(d_iVector1Data, d_iVector2Data, dimSeparatore);	
/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	std::chrono::system_clock::time_point end = std::chrono::high_resolution_clock::now();
	*elapsedDivMul += std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
#endif
*/

		#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &timeDiv, start, stop );
		*elapsedDivMul += timeDiv * 1000000;
		//totalDiv+=timeDiv;
		#endif

/*
		#if DEBUG_FLAG
		//if(debug) {
			cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); 
		//}
		#endif
*/

		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error11: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		/*
		// copy vector from device to host to check errors
		hipMemcpy(h_iVector2Data, d_iVector1Data, sizeof(double)*n, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
		// print the CUDA error message and exit
		printf("\nCUDA error12: %s\n", hipGetErrorString(error));
		exit(-1);
		}

		double prec=h_iVector1Data[0];
		for(int i=0; i < n ; i++)	{
		if(prec != h_iVector1Data[i]) printf("\nERRORE:  h_iVector1Data[%d-1]=%f != h_iVector1Data[%d]=%f\n",i,prec,i, h_iVector1Data[i]);
		prec=h_iVector1Data[i];
		}
		if(prec != 2) printf("\nERRORE:  prec=%f\n",prec);
		*/

		hipFree(d_iVector2Data);	//ALE
		// ALE questo non devo cancellarlo perché c'è il nuovo separatore salvato
		//free(h_iVector2Data);													//ALE
		// ********************** inizio moltiplicazione di h_MatrixData *********************
		// allocate device memory and data	
		double *d_MatrixData = NULL;
		size_t *d_MatrixIndex = NULL;
		hipMalloc((void **) &d_MatrixData, bytesDataMatrix);
		hipMalloc((void **) &d_MatrixIndex, bytesIndexMatrix);

		// copy data directly to device memory 		
		// se d_iVector1Data è piccolo si può provare a dichiararlo come costante (non dall'inizio, perchè prima è modificato)
		// analogamente per d_iVector1Data nella fase precedente
		hipMemcpy(d_MatrixData, h_MatrixData, bytesDataMatrix, hipMemcpyHostToDevice);
		hipMemcpy(d_MatrixIndex, h_MatrixIndex, bytesIndexMatrix, hipMemcpyHostToDevice);
		getNumBlocksAndThreadsDivMult(size, maxThreadsMult, numBlocks, numThreads);
		hipDeviceSynchronize();
		// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
		dim3 dimBlockMult(numThreads, 1, 1);
		// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
		dim3 dimGridMult(numBlocks, 1, 1);
/*
		#if DEBUG_FLAG
		//if(debug) { 
			cudaPrintfInit(); 
			printf("\n MULT:\n %d dimGrid\n %d dimBlock\n",dimGridMult.x, dimBlockMult.x ); 
		//}
		#endif
*/


		#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
		hipEventRecord( start, 0 );
		#endif

/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	begin = std::chrono::high_resolution_clock::now();
#endif
*/
		kernelMultMatrixVector<<< dimGridMult, dimBlockMult >>>(d_MatrixData, d_MatrixIndex, d_iVector1Data, n, size, dimCricca);
/*
#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
	end = std::chrono::high_resolution_clock::now();
	*elapsedDivMul += std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count();
#endif
*/
/*
		#if DEBUG_FLAG
		//if(debug) { 
			cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); 
		//}
		#endif
*/
		// risultato atteso moltiplicazione: 4			// ALE

		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error25: %s\n", hipGetErrorString(error));
			exit(-1);
		}

// se c'è viene incluso nel conteggio del tempo
#if NORMALIZZA_AD_OGNI_PASSO		
		// FASE DI NORMALIZZAZIONE
		/* PROVA SENZA NORMALIZZAZIONE */
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		double sum; // TODO dopo metto d_sum = hipMalloc(...)
		hipblasDasum(handle, dimCricca, d_MatrixData, 1, &sum);
		//printf("la somma degli elementi: %f\n", sum);
		if (sum != 1.0) {
			// normalizzo solo se necessario
			sum = 1.0 / sum;
			hipblasDscal(handle, dimCricca, &sum, d_MatrixData, 1);
			//getchar();
		}
		hipblasDestroy(handle);
		/* */
		//
#endif


		#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
		float timeMult = 0;
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &timeMult, start, stop );
		//totalMult+=timeMult;
		*elapsedDivMul += timeMult * 1000000;
		#endif

		// copy final matrix from device to host
		hipMemcpy(h_MatrixData, d_MatrixData, bytesDataMatrix, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error27: %s\n", hipGetErrorString(error));
			exit(-1);
		}

		/*
		// check matrix results
		int precM=h_MatrixData[0];
		for(int i=0; i < size ; i++)	{
			if(precM != h_MatrixData[i]) printf("\nERRORE:  h_MatrixData[%d-1]=%f != h_MatrixData[%d]=%f\n",i,precM,i, h_MatrixData[i]);
			precM=h_MatrixData[i];
		}
		//assert (m==prec);
		if(precM != 4) printf("\nERRORE:  prec=%f\n",precM);
		*/

		// 64000/8 = 8000 elementi nella constant memory
		#if !TEMPO_COMPLESSIVO && CONSIDERA_MARGINALIZZAZIONE_E_SCATTERING_DIVISE && !CONSIDERA_TRASFERIMENTI_MEMORIA
		hipEventDestroy( start );													//ALE
		hipEventDestroy( stop );													//ALE
		#endif

		//totalMult+=timeMult;	

		//for (size_t l = 0; l < dimCricca; l++)
		//	printf("el[%d] = %f\n", l, h_MatrixData[l]);



		//free(h_iVector1Data);											//ALE
		//free(h_iVector2Data);
		//free(h_MatrixData);
		//free(h_MatrixIndex);
		hipFree(d_iVector1Data);													//ALE
		//hipFree(d_iVector2Data);													//ALE
		hipFree(d_MatrixData);													//ALE
		hipFree(d_MatrixIndex);													//ALE
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error29: %s\n", hipGetErrorString(error));
			exit(-1);
		}
	//} 
	/*
	if(bestAverageTimeDiv > (totalDiv/100) && (!debug)) { bestAverageTimeDiv = (totalDiv/100); bestMaxThreadsDiv = maxThreadsDiv; 
	printf("bestPartialAverageTimeDiv: %f\n bestPartialMaxThreads = %d\n", bestAverageTimeDiv, bestMaxThreadsDiv);
	}
	if(bestAverageTimeMult > (totalMult/100) && (!debug)) { bestAverageTimeMult = (totalMult/100); bestMaxThreadsMult = maxThreadsMult;
	printf("bestPartialAverageTimeMult: %f\n bestPartialMaxThreads = %d\n", bestAverageTimeMult, bestMaxThreadsMult);
	}		
//}
if(!debug){
	printf("bestAverageTimeDiv: %f\n bestMaxThreadsDiv = %d\n", bestAverageTimeDiv, bestMaxThreadsDiv);
	printf("bestAverageTimeMult: %f\n bestMaxThreadsMult = %d\n", bestAverageTimeMult, bestMaxThreadsMult);
}
*/
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// TUTTO INSIEME
//void margAndScatt(size_t sizeCriccaPOW2, size_t sizeSeparatorePOW2, double *tabellaCricca, size_t *tabellaIndiciCricca, size_t dimCricca, size_t dimSeparatore, // dati per marginalization
//			size_t sizeCriccaMulPOW2, double* tabellaSeparatore, double* tabellaCriccaMul, size_t* tabellaIndiciCriccaMul, size_t dimCriccaMul // dati per scattering
//			) {
//
//	double* fiStarOnGpu = marginalizationBigN(sizeCriccaPOW2, sizeSeparatorePOW2, tabellaCricca, tabellaIndiciCricca, dimCricca, dimSeparatore);
//	scattering(sizeCriccaMulPOW2,  // dimTabCricca POW2
//			sizeSeparatorePOW2, // dimTabSep POW2
//			fiStarOnGpu, // double *h_iVector2Data qui ci va il device_v1data
//			tabellaSeparatore, // FI
//			tabellaCriccaMul, // PSI da aggiornare
//			tabellaIndiciCriccaMul, // Indexig of PSI da aggiornare
//
//			dimCriccaMul, // dimensione vera tabCricca
//			dimSeparatore // dim vera tabSe,p
//			);
//}




///////////////////
void selectGPU()
{
	hipDeviceProp_t deviceProp;
    deviceProp.major = 1;

		// da fare nel caso dei double (perchè necessitano di maggiori risorse), altrimenti sarebbero state settate a 1 e 10
	    deviceProp.minor = 3;
	    int minimumComputeVersion = 13;

	    int dev = findCudaDevice();

	    hipGetDeviceProperties(&deviceProp, dev);

	    if ((deviceProp.major * 10 + deviceProp.minor) >= minimumComputeVersion)
	    {
	        printf("Using Device %d: %s\n\n", dev, deviceProp.name);
	        hipSetDevice(dev);
	    }
	    else
	    {
	        printf("Error: the selected device does not support the minimum compute capability of %d.%d.\n\n",
	               minimumComputeVersion / 10, minimumComputeVersion % 10);

	        hipDeviceReset();
	        exit(EXIT_FAILURE);
	    }
}

void resetGPU()
{
	hipDeviceReset();
}
///////////////////

double* recuperaDaGPU(double* tabella, size_t sizeTabella)
{
	size_t bytesTabella = sizeof(double) * sizeTabella;
	double* tabellaCPU = (double*) malloc(bytesTabella);
	hipMemcpy(tabellaCPU, tabella, bytesTabella, hipMemcpyDeviceToHost);
	hipFree(tabella);

	return tabellaCPU;
}

#endif
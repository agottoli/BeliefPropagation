#include "hip/hip_runtime.h"
#ifndef MYLIBKERNELBIGN_CU
#define MYLIBKERNELBIGN_CU

#include <limits.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "myLibKernelBigN.h"

#include "myLib.h"

const bool debug = false;

#ifndef SIZE_MAX
#define SIZE_MAX ((size_t)-1)
#endif

void getNumBlocksAndThreadsBigN(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}

__global__ void
reduce1StepBigN(double *g_idata, double *g_odata, size_t *d_iIndexData, size_t n, size_t halfN, size_t nArray, bool debug)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;

	// somma elementi della prima metà con elementi della seconda metà
	int index1 = d_iIndexData[i+halfN];
	int index2 = d_iIndexData[i];
		
    if (i < halfN){ // sta sempre dentro all'if
		//g_odata[i] = g_idata[d_iIndexData[i]] + g_idata[d_iIndexData[i+halfN]];
		double mySum = (index1 != SIZE_MAX) ? g_idata[index1] : 0;
		g_odata[i] = (index2 != SIZE_MAX) ? g_idata[index2] + mySum : mySum;
			if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
	}
}

__global__ void														// non so se passarlo come parametro o farlo calcolare alle theads... TESTARE
reduce2StepBigN(double *g_idata, double *g_odata, size_t n, size_t halfN, size_t nArray, bool debug)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	unsigned const int i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;
    //double mySum = (i < half) ? g_idata[i] : 0;
	// somma elementi della prima metà con elementi della seconda metà
    if (i < halfN){ // sta sempre dentro all'if
        //mySum += g_idata[i+blockDim.x];
		//double mySum = g_idata[i] + g_idata[i+blockDim.x];
		g_odata[i] = g_idata[i] + g_idata[i+halfN];
			if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
	}
}



double *reduceBigNArray(size_t  n,
				  	size_t nArray,
                 	int  numThreads,
                  	int  numBlocks,
                  	int  maxThreads,
                  	//int  cpuFinalThreshold,
                  	double *h_odata, // si può togliere ??
                  	double *d_idata,
                  	double *d_odata,
					size_t *d_iIndexData,
					size_t dimRisultato)
{
	bool needReadBack = true;
	
    hipDeviceSynchronize();
    	// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
    	dim3 dimBlock(numThreads, 1, 1);
    	// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
	    dim3 dimGrid(numBlocks, 1, 1);

		hipError_t error;
		
		//size_t dimResult = nArray * sizeof(double);
		size_t dimResult = dimRisultato * sizeof(double);

		double *gpu_result = (double *) malloc(dimResult);
		// ALE_DOMANDA
		// SERVE??? non viene sovrascritto poi dalla cudaMemCpy???
		for (int i=0; i < dimRisultato; i++) gpu_result[i]=0;		
		//

		if(debug) { 
			cudaPrintfInit();
				// execute the kernel
				printf("\nSTART esecuzione:\n %d dimGrid\n %d dimBlock\n",dimGrid.x, dimBlock.x ); }
		reduce1StepBigN<<< dimGrid, dimBlock>>>(d_idata, d_odata, d_iIndexData, n, n>>1, nArray, debug);
		if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
		/*
		unsigned int dimResultCHECK = (n>>1) * sizeof(double);
		double *gpu_resultCHECK = (double *) malloc(dimResultCHECK);
		hipMemcpy(gpu_resultCHECK, d_odata, sizeof(double)*(n>>1), hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error0: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
		double prec=gpu_resultCHECK[0];
		for(int i=0; i < nArray ; i++)	{
			if(prec != gpu_resultCHECK[i]) printf("\n---0 ERRORE:  gpu_resultCHECK[%d-1]=%f != gpu_resultCHECK[%d]=%f",i,prec,i, gpu_resultCHECK[i]);
			prec=gpu_resultCHECK[i];
		//    printf("\nGPU result = %f\n", gpu_result[i]);
		}
		int j=0;
		
        printf("\nSTART %d GPU result       \t= \t %f\n", j, prec);
        */


		for(size_t nElements = n>>1; nElements > nArray; nElements>>=1){
			//sleep(1);
			
			int threads = 0, blocks = 0;
            

			getNumBlocksAndThreadsBigN(nElements, maxThreads, blocks, threads);
			
			dim3 dimBlock(threads, 1, 1);
			dim3 dimGrid(blocks, 1, 1);
			
			if(debug) printf("\n%d elementi:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); 
	
			if(debug) { 
				cudaPrintfInit();
					// execute the kernel
					printf("\n%d esecuzione:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); }
			reduce2StepBigN<<< dimGrid, dimBlock>>>(d_odata, d_odata, nElements, nElements>>1, nArray, debug);
			if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
			
        }
        
		if (needReadBack)
	    {
	        // copy final sum from device to host
          //hipMemcpy(gpu_result, d_odata, sizeof(double)*nArray, hipMemcpyDeviceToHost);
			hipMemcpy(gpu_result, d_odata, dimResult, hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error2: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
	    }
	return gpu_result;   
}

double* runBigN(size_t size, size_t nArray, double *h_idata, size_t *h_iIndexData, size_t dimInput, size_t dimRisultato){
	// ALE_GOT
	//size = 1048576;
	//nArray = 16384;
	//

	//int size = 1<<24; // 1<<24;    // number of elements to reduce -> default:  16777216				// ALE
	//int nArray = 1<<4;	// m = 16384 
	//int m = 1<<20; // 4 7 10 14 17 20 
	size_t m = size / nArray;																					// ALE
	//size = 1<<16;																						// ALE
	//nArray = 1<<20;																					// ALE
    int maxThreads = 256;				// number of threads per block
    int cpuFinalThreshold = 1;
    /*  STAMPE */
	printf("%d elements\n", size);
    printf("%d nArray\n", nArray);
	printf("%d nArrayInput\n", dimInput);
	printf("%d nArrayRisultato\n", dimRisultato);
    printf("%d m\n", m);
    printf("%d maxThreads\n", maxThreads);
	/* */

	double *gpu_result;	

	if (m == 1) {
		gpu_result = (double *) malloc(dimRisultato * sizeof(double));
		for (size_t i = 0; i < dimInput; i++)
			gpu_result[i] = h_idata[i];
		return gpu_result;
	}
	
	float total = 0.0;

	//for(int i =0; i<100; i++){

        // create input
	size_t bytesValuesInput = dimInput * sizeof(double);										// ALE
        // ALE_GOT
		//double *
		//	h_idata = (double *) malloc(bytesValuesInput);										// ALE
		//
		size_t bytesIndexInput = size * sizeof(size_t);										// ALE
		// ALE_GOT
		//size_t *
		//	h_iIndexData = (size_t *) malloc(bytesIndexInput);										// ALE
		//
		// ALE_GOT
		//for (size_t i=0; i<size; i++) { h_idata[i] = 1; h_iIndexData[i]=i; }							// ALE
		//

       	int numBlocks = 0;
        int numThreads = 0;
		getNumBlocksAndThreadsBigN(size, maxThreads, numBlocks, numThreads);
       
	    if (numBlocks == 1) cpuFinalThreshold = 1;
       
        // allocate mem for the result on host side
		size_t bytesOutput = sizeof(double)*(size/2);											// ALE
        double *h_odata = (double *) malloc(bytesOutput);
        if(debug) printf("%d blocks\n", numBlocks);
       
		// allocate device memory and data
        double *d_idata = NULL;
		size_t *d_iIndexData = NULL;
        double *d_odata = NULL;
	
	

        hipMalloc((void **) &d_idata, bytesValuesInput);
        hipMalloc((void **) &d_iIndexData, bytesIndexInput);
        hipMalloc((void **) &d_odata, bytesOutput);

        // copy data directly to device memory
        hipMemcpy(d_idata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);

		// ALE_GOT
		/*
		for (size_t l = 0; l < dimInput; l++)
				printf("%f ", h_idata[l]);
			printf("\n");
			for (size_t l = 0; l < size; l++)
				printf("%d ", h_iIndexData[l]);
		*/
		//	getchar();
		
		//


        hipMemcpy(d_iIndexData, h_iIndexData, bytesIndexInput, hipMemcpyHostToDevice);
	    //printf("\nbytesOutput=%d\n",bytesOutput);
        //hipMemcpy(d_odata, h_idata, bytesOutput, hipMemcpyHostToDevice);			//NON SERVE!!!

		// per i test si può inserire un warm-up
			hipEvent_t start, stop;
			float time;	// GPU TIME
			hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
			hipEventCreate(&stop);	
		
		
			hipEventRecord( start, 0 );
		
	        //double *gpu_result;																		// ALE è lungo nArray

	        gpu_result = reduceBigNArray(size, nArray, numThreads, numBlocks, maxThreads,
										h_odata, d_idata, d_odata, d_iIndexData, dimRisultato);


			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &time, start, stop );
			total+=time;										// ALE 
			//printf ("\nTime for the kernel: %f ms\n", time);											// ALE 
			double prec=gpu_result[0];
			/*for(int i=0; i < nArray ; i++)	{
				if(prec != gpu_result[i]) {printf("\nERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); exit(-1);}
				prec=gpu_result[i];
			//    printf("\nGPU result = %f\n", gpu_result[i]);
			}*/
//			assert (m==prec);
			hipEventDestroy( start );													// ALE
			hipEventDestroy( stop );													//ALE
			
			
	        //printf("\nGPU result       \t= \t %f\n", prec);
			assert ((size % nArray) == 0);
			//printf("risultato atteso \t= \t %d\n", size / nArray);


			//free(gpu_result);	    
			//free(h_iIndexData);    
	        //free(h_idata);													//ALE
	        //free(h_odata);													//ALE
			hipFree(d_idata);													//ALE
	        hipFree(d_odata);													//ALE
			hipFree(d_iIndexData);
		//} 
		printf("average time: %f\n", (total/100));

		// ALE_GOT
		return gpu_result;
		//


		
}



///////////////////
void selectGPU()
{
	hipDeviceProp_t deviceProp;
    deviceProp.major = 1;

		// da fare nel caso dei double (perchè necessitano di maggiori risorse), altrimenti sarebbero state settate a 1 e 10
	    deviceProp.minor = 3;
	    int minimumComputeVersion = 13;

	    int dev = findCudaDevice();

	    hipGetDeviceProperties(&deviceProp, dev);

	    if ((deviceProp.major * 10 + deviceProp.minor) >= minimumComputeVersion)
	    {
	        printf("Using Device %d: %s\n\n", dev, deviceProp.name);
	        hipSetDevice(dev);
	    }
	    else
	    {
	        printf("Error: the selected device does not support the minimum compute capability of %d.%d.\n\n",
	               minimumComputeVersion / 10, minimumComputeVersion % 10);

	        hipDeviceReset();
	        exit(EXIT_FAILURE);
	    }
}

void resetGPU()
{
	hipDeviceReset();
}
///////////////////

#endif
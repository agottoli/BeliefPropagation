#include "hip/hip_runtime.h"
#ifndef MYLIBKERNELBIGN_CU
#define MYLIBKERNELBIGN_CU

#include <limits.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "myLibKernelBigN.h"

#include "myLib.h"

#include "hipblas.h"
#include "windows.h"

const bool debug = false;

#ifndef SIZE_MAX
#define SIZE_MAX ((size_t)-1)
#endif

//#ifndef zeroALE
//#define zeroALE 0.0
//#endif

//////////////////////////////////////////// RIDUZIONE ==> MARGINALIZATION //////////////////////////////////////////////////////

void getNumBlocksAndThreadsBigN(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}

__global__ void
reduce1StepBigN(double *g_idata, double *g_odata, size_t *d_iIndexData, size_t n, size_t halfN, size_t nArray, bool debug)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;

	// somma elementi della prima metà con elementi della seconda metà
	int index1 = d_iIndexData[i+halfN];
	int index2 = d_iIndexData[i];
		
    if (i < halfN){ // sta sempre dentro all'if
		//g_odata[i] = g_idata[d_iIndexData[i]] + g_idata[d_iIndexData[i+halfN]];
		double mySum = (index1 != SIZE_MAX) ? g_idata[index1] : 0;
		g_odata[i] = (index2 != SIZE_MAX) ? g_idata[index2] + mySum : mySum;
			if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
	}
}

__global__ void														// non so se passarlo come parametro o farlo calcolare alle theads... TESTARE
reduce2StepBigN(double *g_idata, double *g_odata, size_t n, size_t halfN, size_t nArray, bool debug)//, unsigned int fraction)
{	
	// i scorre la prima metà dei dati in input
	unsigned const int i = blockIdx.x*blockDim.x + threadIdx.x;
	//unsigned const int halfN = n>>1;
    //double mySum = (i < half) ? g_idata[i] : 0;
	// somma elementi della prima metà con elementi della seconda metà
    if (i < halfN){ // sta sempre dentro all'if
        //mySum += g_idata[i+blockDim.x];
		//double mySum = g_idata[i] + g_idata[i+blockDim.x];
		g_odata[i] = g_idata[i] + g_idata[i+halfN];
			if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) 
				cuPrintf ("CUPRINTF 4- g_odata[%d*%d + %d] = %f \n", blockIdx.x, nArray, threadIdx.x, g_odata[threadIdx.x]);
	}
}



void reduceBigNArray(size_t  n,
				  	size_t nArray,
                 	int  numThreads,
                  	int  numBlocks,
                  	int  maxThreads,
                  	//int  cpuFinalThreshold,
                  	double *h_odata, // si può togliere ??
                  	double *d_idata,
                  	double *d_odata,
					size_t *d_iIndexData,
					size_t dimRisultato)
{
	bool needReadBack = true;
	
    hipDeviceSynchronize();
    	// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
    	dim3 dimBlock(numThreads, 1, 1);
    	// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
	    dim3 dimGrid(numBlocks, 1, 1);

		hipError_t error;
		
		//size_t dimResult = nArray * sizeof(double);
		size_t dimResult = dimRisultato * sizeof(double);

		// ALE 2014-02-23
		/*
		double *gpu_result = (double *) malloc(dimResult);
		// ALE_DOMANDA
		// SERVE??? non viene sovrascritto poi dalla cudaMemCpy???
		for (int i=0; i < dimRisultato; i++) gpu_result[i]=0;		
		//
		*/
		if(debug) { 
			cudaPrintfInit();
				// execute the kernel
				printf("\nSTART esecuzione:\n %d dimGrid\n %d dimBlock\n",dimGrid.x, dimBlock.x ); }
		reduce1StepBigN<<< dimGrid, dimBlock>>>(d_idata, d_odata, d_iIndexData, n, n>>1, nArray, debug);
		if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
		/*
		unsigned int dimResultCHECK = (n>>1) * sizeof(double);
		double *gpu_resultCHECK = (double *) malloc(dimResultCHECK);
		hipMemcpy(gpu_resultCHECK, d_odata, sizeof(double)*(n>>1), hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error0: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	
		double prec=gpu_resultCHECK[0];
		for(int i=0; i < nArray ; i++)	{
			if(prec != gpu_resultCHECK[i]) printf("\n---0 ERRORE:  gpu_resultCHECK[%d-1]=%f != gpu_resultCHECK[%d]=%f",i,prec,i, gpu_resultCHECK[i]);
			prec=gpu_resultCHECK[i];
		//    printf("\nGPU result = %f\n", gpu_result[i]);
		}
		int j=0;
		
        printf("\nSTART %d GPU result       \t= \t %f\n", j, prec);
        */


		for(size_t nElements = n>>1; nElements > nArray; nElements>>=1){
			//sleep(1);
			
			int threads = 0, blocks = 0;
            

			getNumBlocksAndThreadsBigN(nElements, maxThreads, blocks, threads);
			
			dim3 dimBlock(threads, 1, 1);
			dim3 dimGrid(blocks, 1, 1);
			
			if(debug) printf("\n%d elementi:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); 
	
			if(debug) { 
				cudaPrintfInit();
					// execute the kernel
					printf("\n%d esecuzione:\n %d dimGrid\n %d dimBlock\n",nElements,dimGrid.x, dimBlock.x ); }
			reduce2StepBigN<<< dimGrid, dimBlock>>>(d_odata, d_odata, nElements, nElements>>1, nArray, debug);
			if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
			
        }
        
		if (needReadBack)
	    {
	        // copy final sum from device to host
          //hipMemcpy(gpu_result, d_odata, sizeof(double)*nArray, hipMemcpyDeviceToHost);
			
			// ALE CUDA MEM CPY lo faccio dopo visto che lo unisco allo scattering
			/*hipMemcpy(gpu_result, d_odata, dimResult, hipMemcpyDeviceToHost);
		  error = hipGetLastError();
		  if(error != hipSuccess)
		  {
		    // print the CUDA error message and exit
		    printf("\nCUDA error2: %s\n", hipGetErrorString(error));
		    exit(-1);
		  }	*/
	    }
		// ALE 2014-02-23
	/*
	return gpu_result;  
	*/

}

//                   dimTabCricca POW2, dimTabSep POW2, tabCricca,     tabIndiciCricca,   dimensione vera tabCricca, dim vera tabSep
double* marginalizationBigN(size_t size, size_t nArray, double *h_idata, size_t *h_iIndexData, size_t dimInput, size_t dimRisultato){
	// ALE_GOT
	//size = 1048576;
	//nArray = 16384;
	//

	//int size = 1<<24; // 1<<24;    // number of elements to reduce -> default:  16777216				// ALE
	//int nArray = 1<<4;	// m = 16384 
	//int m = 1<<20; // 4 7 10 14 17 20 
	size_t m = size / nArray;																					// ALE
	//size = 1<<16;																						// ALE
	//nArray = 1<<20;																					// ALE
    int maxThreads = 256;				// number of threads per block
    int cpuFinalThreshold = 1;
    /*  STAMPE * /
	printf("%d elements\n", size);
    printf("%d nArray\n", nArray);
	printf("%d nArrayInput\n", dimInput);
	printf("%d nArrayRisultato\n", dimRisultato);
    printf("%d m\n", m);
    printf("%d maxThreads\n", maxThreads);
	/ * */

	//double *gpu_result;	

	
	if (m == 1) {
		// ALE 2014-02-23
		/*
		gpu_result = (double *) malloc(dimRisultato * sizeof(double));
		for (size_t i = 0; i < dimInput; i++)
			gpu_result[i] = h_idata[i];
		return gpu_result;
		*/
		double *d_odata = NULL;
		size_t bytesValuesInput = sizeof(double)*dimInput;
		hipMalloc((void **) &d_odata, bytesValuesInput);
		// copy data directly to device memory
        hipMemcpy(d_odata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);
		return d_odata;
	}

	
	float total = 0.0;

	//for(int i =0; i<100; i++){

        // create input
	size_t bytesValuesInput = dimInput * sizeof(double);										// ALE
        // ALE_GOT
		//double *
		//	h_idata = (double *) malloc(bytesValuesInput);										// ALE
		//
		size_t bytesIndexInput = size * sizeof(size_t);										// ALE
		// ALE_GOT
		//size_t *
		//	h_iIndexData = (size_t *) malloc(bytesIndexInput);										// ALE
		//
		// ALE_GOT
		//for (size_t i=0; i<size; i++) { h_idata[i] = 1; h_iIndexData[i]=i; }							// ALE
		//

       	int numBlocks = 0;
        int numThreads = 0;
		getNumBlocksAndThreadsBigN(size, maxThreads, numBlocks, numThreads);
       
	    if (numBlocks == 1) cpuFinalThreshold = 1;
       
        // allocate mem for the result on host side
		size_t bytesOutput = sizeof(double)*(size/2);											// ALE
        double *h_odata = (double *) malloc(bytesOutput);
        if(debug) printf("%d blocks\n", numBlocks);
       
		// allocate device memory and data
        double *d_idata = NULL;
		size_t *d_iIndexData = NULL;
        double *d_odata = NULL;
	
	

        hipMalloc((void **) &d_idata, bytesValuesInput);
        hipMalloc((void **) &d_iIndexData, bytesIndexInput);
        hipMalloc((void **) &d_odata, bytesOutput);

        // copy data directly to device memory
        hipMemcpy(d_idata, h_idata, bytesValuesInput, hipMemcpyHostToDevice);

		// ALE_GOT
		/*
		for (size_t l = 0; l < dimInput; l++)
				printf("%f ", h_idata[l]);
			printf("\n");
			for (size_t l = 0; l < size; l++)
				printf("%d ", h_iIndexData[l]);
		*/
		//	getchar();
		
		//


        hipMemcpy(d_iIndexData, h_iIndexData, bytesIndexInput, hipMemcpyHostToDevice);
	    //printf("\nbytesOutput=%d\n",bytesOutput);
        //hipMemcpy(d_odata, h_idata, bytesOutput, hipMemcpyHostToDevice);			//NON SERVE!!!

		// per i test si può inserire un warm-up
			hipEvent_t start, stop;
			float time;	// GPU TIME
			hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
			hipEventCreate(&stop);	
		
		
			hipEventRecord( start, 0 );
		
	        //double *gpu_result;																		// ALE è lungo nArray
			
			//ALE 2014-02-23
			/*
	        gpu_result = 
			*/
			reduceBigNArray(size, nArray, numThreads, numBlocks, maxThreads,
										h_odata, d_idata, d_odata, d_iIndexData, dimRisultato);


			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &time, start, stop );
			total+=time;										// ALE 
			//printf ("\nTime for the kernel: %f ms\n", time);											// ALE 
			//double prec=gpu_result[0];
			/*for(int i=0; i < nArray ; i++)	{
				if(prec != gpu_result[i]) {printf("\nERRORE:  gpu_result[%d-1]=%f != gpu_result[%d]=%f",i,prec,i, gpu_result[i]); exit(-1);}
				prec=gpu_result[i];
			//    printf("\nGPU result = %f\n", gpu_result[i]);
			}*/
//			assert (m==prec);
			hipEventDestroy( start );													// ALE
			hipEventDestroy( stop );													//ALE
			
			
	        //printf("\nGPU result       \t= \t %f\n", prec);
			assert ((size % nArray) == 0);
			//printf("risultato atteso \t= \t %d\n", size / nArray);


			//free(gpu_result);	    
			//free(h_iIndexData);    
	        //free(h_idata);													//ALE
	        //free(h_odata);													//ALE
			hipFree(d_idata);													//ALE
	        //hipFree(d_odata);													//ALE
			hipFree(d_iIndexData);
		//} 
		/*
		printf("average time: %f\n", (total/100));
		*/
		// ALE_GOT
		//return gpu_result;
			return d_odata;
		//


		
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////// MOLTIPLICAZIONI ==> SCATTERING /////////////////////////////////////////////////////

void getNumBlocksAndThreadsDivMult(size_t n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device;
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = n / threads; //(n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}



// (possibile ottimizzazione: far si che ogni blocco faccia le operazioni sul doppio dei dati
// per avere la coalescenza andrebbe fatto non a livello di thread (th0 legge dato0 e dato1)
// ma th0 legge dato 0 e dato 0+dimBlocco
__global__ void
kernelDivVector(double *g_iVector1Data, double *g_iVector2Data, size_t n, const bool debug)
{	
//	if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 1- \n");
	size_t i = blockIdx.x*(blockDim.x) + threadIdx.x;
	if (i < n)
		// ALE
		g_iVector1Data[i] = (g_iVector2Data[i] > 0.0) ? g_iVector1Data[i] / g_iVector2Data[i] : 0; // zeroALE = 0.0
}

/*void getNumBlocksAndThreadsMultMatrixVector(int n, int maxThreads, int &blocks, int &threads){
        //get device capability, to avoid block/grid size excceed the upbound
	    hipDeviceProp_t prop;
	    int device; 
		hipGetDevice(&device);
		hipGetDeviceProperties(&prop, device);
		
		threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
		blocks = n/threads; //(n + (threads * 2 - 1)) / (threads * 2);
	    
	    if ((float)threads*blocks > (float)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
	    {
	        printf("size is too large, please choose a smaller number!\n");
	    }
		
		if (blocks > prop.maxGridSize[0])
	    {
	        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
	               blocks, prop.maxGridSize[0], threads*2, threads);
	        blocks /= 2;
	        threads *= 2;
	    }
}*/

	__global__ void
kernelMultMatrixVector(double *d_MatrixData, size_t *d_MatrixIndex, double * g_iVector1Data, size_t n, size_t size, const bool debug){
	size_t i = blockIdx.x*(blockDim.x) + threadIdx.x;
	size_t index = d_MatrixIndex[i];
	if (index != SIZE_MAX)
		d_MatrixData[index] *= g_iVector1Data[i % n];
	
	//if(debug && (threadIdx.x == 0 || threadIdx.x == (blockDim.x-1))&&(blockIdx.x==0 || blockIdx.x==gridDim.x-1)) cuPrintf ("CUPRINTF 3- d_MatrixData[%d] = %f \n", index, d_MatrixData[index]);
}

//           dimTabCricca POW2, dimTabSep POW2, tabCricca, tabIndiciCricca,        dimensione vera tabCricca, dim vera tabSep
void scattering(size_t size,  // dimTabCricca POW2
			size_t n, // dimTabSep POW2
			double *d_iVector1Data, // double *h_iVector2Data qui ci va il device_v1data
			double *h_iVector2Data, // FI
			double *h_MatrixData, // PSI da aggiornare
			size_t *h_MatrixIndex, // Indexig of PSI da aggiornare

			size_t dimCricca, // dimensione vera tabCricca
			size_t dimSeparatore // dim vera tabSep
			) {
	// TESTA CON NVPROF
	//unsigned int n = 1<<12;//19;	
	//unsigned int m = 1<<12;//5;
	//unsigned int size = m*n;

	size_t m = size / n;

	//int maxThreadsDiv = 512;				/// ATTENZIONE, QUANDO SI FANNO I TEST BISOGNA VARIARE ENTRAMBI I VALORI!!!
	/*
	printf("%d n\n", n);
	printf("%d m\n", m);
	printf("%d size\n", size);
	printf("%d dimSep\n", dimSeparatore);
	printf("%d dimCricca\n", dimCricca);
	*/
	hipError_t error;

	float bestAverageTimeMult = 100.0;
	float bestAverageTimeDiv = 100.0;
	unsigned int bestMaxThreadsDiv = 0;
	unsigned int bestMaxThreadsMult = 0;

	
	//for (int maxThreads = 32; maxThreads < 1024; maxThreads*=2) {
	// ALE
	int maxThreads = 256;

	unsigned int maxThreadsDiv = maxThreads;
	unsigned int maxThreadsMult = maxThreads;

	float totalDiv = 0.0;
	float totalMult = 0.0;
		
	//for(int i =0; i<100; i++){
	
		float timeDiv=0;																						// GPU TIME
		// create input
		// BISOGNA TOGLIERE GLI INDICI DALL'ARRAY, (METTERE L'ARRAY NELLA CONSTANT MEMORY)...
		size_t bytesVector = n * sizeof(double);										// ALE
		// FI* (il risultato della divisione lo salva nel g_ di questo)
		// passo direttamente il g_
		//double *h_iVector1Data = (double *) malloc(bytesVector);									// ALE
		// FI
		//double *h_iVector2Data = (double *) malloc(bytesVector);									// ALE

		size_t bytesDataMatrix = dimCricca * sizeof(double);										// ALE
		//double *h_MatrixData = (double *) malloc(bytesDataMatrix);									// ALE
		size_t bytesIndexMatrix = size * sizeof(size_t);									// ALE
		//unsigned int *h_MatrixIndex = (unsigned int *) malloc(bytesIndexMatrix);					// ALE


		/*
		for (unsigned int i=0; i<n; i++) { h_iVector1Data[i] = 10; h_iVector2Data[i] = 5; } 	// risultato atteso divisione: 2	// ALE
		for (unsigned int i=0; i<size; i++) { 
			h_MatrixIndex[i] = i; 
			h_MatrixData[i] = 2; 
		} // risultato atteso moltiplicazione: 4			// ALE
		*/

		int numBlocks = 0;
		int numThreads = 0;
		getNumBlocksAndThreadsDivMult(n, maxThreadsDiv, numBlocks, numThreads);

		assert(n%numThreads==0);

		// allocate device memory and data
		//double *d_iVector1DataP = NULL;
		double *d_iVector2Data = NULL;
		//hipMalloc((void **) &d_iVector1DataP, bytesVector);
		hipMalloc((void **) &d_iVector2Data, bytesVector);

		// copy data directly to device memory
		//hipMemcpy(d_iVector1Data, h_iVector1Data, bytesVector, hipMemcpyHostToDevice);
		hipMemcpy(d_iVector2Data, h_iVector2Data, bytesVector, hipMemcpyHostToDevice);

		// ALE adesso il valore di h_iVector2Data è sul device e nell'host non mi serve più quindi lo rimpiazzo con d_iVector1Data
		// copy vector from device to host to check errors
		hipMemcpy(h_iVector2Data, d_iVector1Data, sizeof(double) * dimSeparatore, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error12: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		//

		hipDeviceSynchronize();
		// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
		dim3 dimBlockDiv(numThreads, 1, 1);
		// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
		dim3 dimGridDiv(numBlocks, 1, 1);

		if(debug) { cudaPrintfInit(); 
		printf("\n DIV:\n %d dimGrid\n %d dimBlock\n",dimGridDiv.x, dimBlockDiv.x ); }

		hipEvent_t start, stop;
		hipEventCreate(&start); // vedi http://docs.nvidia.com/cuda/cuda-c-best-practices-guide/
		hipEventCreate(&stop);	
		hipEventRecord( start, 0 );

		kernelDivVector<<< dimGridDiv, dimBlockDiv >>>(d_iVector1Data, d_iVector2Data, n, debug);			

		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &timeDiv, start, stop );
		//totalDiv+=timeDiv;

		if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error11: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		/*
		// copy vector from device to host to check errors
		hipMemcpy(h_iVector2Data, d_iVector1Data, sizeof(double)*n, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
		// print the CUDA error message and exit
		printf("\nCUDA error12: %s\n", hipGetErrorString(error));
		exit(-1);
		}

		double prec=h_iVector1Data[0];
		for(int i=0; i < n ; i++)	{
		if(prec != h_iVector1Data[i]) printf("\nERRORE:  h_iVector1Data[%d-1]=%f != h_iVector1Data[%d]=%f\n",i,prec,i, h_iVector1Data[i]);
		prec=h_iVector1Data[i];
		}
		if(prec != 2) printf("\nERRORE:  prec=%f\n",prec);
		*/

		hipFree(d_iVector2Data);	//ALE
		// ALE questo non devo cancellarlo perché c'è il nuovo separatore salvato
		//free(h_iVector2Data);													//ALE
		// ********************** inizio moltiplicazione di h_MatrixData *********************
		// allocate device memory and data	
		double *d_MatrixData = NULL;
		size_t *d_MatrixIndex = NULL;
		hipMalloc((void **) &d_MatrixData, bytesDataMatrix);
		hipMalloc((void **) &d_MatrixIndex, bytesIndexMatrix);

		// copy data directly to device memory 		
		// se d_iVector1Data è piccolo si può provare a dichiararlo come costante (non dall'inizio, perchè prima è modificato)
		// analogamente per d_iVector1Data nella fase precedente
		hipMemcpy(d_MatrixData, h_MatrixData, bytesDataMatrix, hipMemcpyHostToDevice);
		hipMemcpy(d_MatrixIndex, h_MatrixIndex, bytesIndexMatrix, hipMemcpyHostToDevice);
		getNumBlocksAndThreadsDivMult(size, maxThreadsMult, numBlocks, numThreads);
		hipDeviceSynchronize();
		// (equivalente) a dim3 dimBlock = dim3(numThreads, 1, 1);
		// le threads avranno quindi un'unica dimensione
		dim3 dimBlockMult(numThreads, 1, 1);
		// (equivalente) a dim3 dimGrid = dim3(numBlocks, 1, 1);
		// griglia di un'unica dimensione
		dim3 dimGridMult(numBlocks, 1, 1);
		if(debug) { cudaPrintfInit(); 
		printf("\n MULT:\n %d dimGrid\n %d dimBlock\n",dimGridMult.x, dimBlockMult.x ); }

		hipEventRecord( start, 0 );

		kernelMultMatrixVector<<< dimGridMult, dimBlockMult >>>(d_MatrixData, d_MatrixIndex, d_iVector1Data, n, size, debug);
		if(debug) { cudaPrintfDisplay (stdout, true); cudaPrintfEnd (); }
		// risultato atteso moltiplicazione: 4			// ALE

		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error25: %s\n", hipGetErrorString(error));
			exit(-1);
		}

		float timeMult = 0;
		hipEventRecord( stop, 0 );
		hipEventSynchronize( stop );
		hipEventElapsedTime( &timeMult, start, stop );
		totalMult+=timeMult;
		
		// FASE DI NORMALIZZAZIONE
		/* PROVA SENZA NORMALIZZAZIONE * /
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		double sum; // TODO dopo metto d_sum = hipMalloc(...)
		hipblasDasum(handle, dimCricca, d_MatrixData, 1, &sum);
		//printf("la somma degli elementi: %f\n", sum);
		if (sum != 1.0) {
			// normalizzo solo se necessario
			sum = 1.0 / sum;
			hipblasDscal(handle, dimCricca, &sum, d_MatrixData, 1);
			//getchar();
		}
		hipblasDestroy(handle);
		/ * */
		//

		// copy final matrix from device to host
		hipMemcpy(h_MatrixData, d_MatrixData, bytesDataMatrix, hipMemcpyDeviceToHost);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error27: %s\n", hipGetErrorString(error));
			exit(-1);
		}

		/*
		// check matrix results
		int precM=h_MatrixData[0];
		for(int i=0; i < size ; i++)	{
			if(precM != h_MatrixData[i]) printf("\nERRORE:  h_MatrixData[%d-1]=%f != h_MatrixData[%d]=%f\n",i,precM,i, h_MatrixData[i]);
			precM=h_MatrixData[i];
		}
		//assert (m==prec);
		if(precM != 4) printf("\nERRORE:  prec=%f\n",precM);
		*/

		// 64000/8 = 8000 elementi nella constant memory

		hipEventDestroy( start );													//ALE
		hipEventDestroy( stop );													//ALE

		totalMult+=timeMult;	

		//free(h_iVector1Data);											//ALE
		//free(h_iVector2Data);
		//free(h_MatrixData);
		//free(h_MatrixIndex);
		hipFree(d_iVector1Data);													//ALE
		//hipFree(d_iVector2Data);													//ALE
		hipFree(d_MatrixData);													//ALE
		hipFree(d_MatrixIndex);													//ALE
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("\nCUDA error29: %s\n", hipGetErrorString(error));
			exit(-1);
		}
	//} 
	/*
	if(bestAverageTimeDiv > (totalDiv/100) && (!debug)) { bestAverageTimeDiv = (totalDiv/100); bestMaxThreadsDiv = maxThreadsDiv; 
	printf("bestPartialAverageTimeDiv: %f\n bestPartialMaxThreads = %d\n", bestAverageTimeDiv, bestMaxThreadsDiv);
	}
	if(bestAverageTimeMult > (totalMult/100) && (!debug)) { bestAverageTimeMult = (totalMult/100); bestMaxThreadsMult = maxThreadsMult;
	printf("bestPartialAverageTimeMult: %f\n bestPartialMaxThreads = %d\n", bestAverageTimeMult, bestMaxThreadsMult);
	}		
//}
if(!debug){
	printf("bestAverageTimeDiv: %f\n bestMaxThreadsDiv = %d\n", bestAverageTimeDiv, bestMaxThreadsDiv);
	printf("bestAverageTimeMult: %f\n bestMaxThreadsMult = %d\n", bestAverageTimeMult, bestMaxThreadsMult);
}
*/
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// TUTTO INSIEME
//void margAndScatt(size_t sizeCriccaPOW2, size_t sizeSeparatorePOW2, double *tabellaCricca, size_t *tabellaIndiciCricca, size_t dimCricca, size_t dimSeparatore, // dati per marginalization
//			size_t sizeCriccaMulPOW2, double* tabellaSeparatore, double* tabellaCriccaMul, size_t* tabellaIndiciCriccaMul, size_t dimCriccaMul // dati per scattering
//			) {
//
//	double* fiStarOnGpu = marginalizationBigN(sizeCriccaPOW2, sizeSeparatorePOW2, tabellaCricca, tabellaIndiciCricca, dimCricca, dimSeparatore);
//	scattering(sizeCriccaMulPOW2,  // dimTabCricca POW2
//			sizeSeparatorePOW2, // dimTabSep POW2
//			fiStarOnGpu, // double *h_iVector2Data qui ci va il device_v1data
//			tabellaSeparatore, // FI
//			tabellaCriccaMul, // PSI da aggiornare
//			tabellaIndiciCriccaMul, // Indexig of PSI da aggiornare
//
//			dimCriccaMul, // dimensione vera tabCricca
//			dimSeparatore // dim vera tabSe,p
//			);
//}




///////////////////
void selectGPU()
{
	hipDeviceProp_t deviceProp;
    deviceProp.major = 1;

		// da fare nel caso dei double (perchè necessitano di maggiori risorse), altrimenti sarebbero state settate a 1 e 10
	    deviceProp.minor = 3;
	    int minimumComputeVersion = 13;

	    int dev = findCudaDevice();

	    hipGetDeviceProperties(&deviceProp, dev);

	    if ((deviceProp.major * 10 + deviceProp.minor) >= minimumComputeVersion)
	    {
	        printf("Using Device %d: %s\n\n", dev, deviceProp.name);
	        hipSetDevice(dev);
	    }
	    else
	    {
	        printf("Error: the selected device does not support the minimum compute capability of %d.%d.\n\n",
	               minimumComputeVersion / 10, minimumComputeVersion % 10);

	        hipDeviceReset();
	        exit(EXIT_FAILURE);
	    }
}

void resetGPU()
{
	hipDeviceReset();
}
///////////////////

#endif